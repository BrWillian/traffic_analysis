#include "hip/hip_runtime.h"
#include <assert.h>
#include <vector>
#include <iostream>
#include "../include/yololayer.h"
#include "../include/utils.h"

namespace Tn
{
    template<typename T>
    void write(char*& buffer, const T& val)
    {
        *reinterpret_cast<T*>(buffer) = val;
        buffer += sizeof(T);
    }

    template<typename T>
    void read(const char*& buffer, T& val)
    {
        val = *reinterpret_cast<const T*>(buffer);
        buffer += sizeof(T);
    }
}

using namespace Yolo;

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin(int classCount, int netWidth, int netHeight, int maxOut, const std::vector<Yolo::YoloKernel>& vYoloKernel)
    {
        mClassCount = classCount;
        mYoloV5NetWidth = netWidth;
        mYoloV5NetHeight = netHeight;
        mMaxOutObject = maxOut;
        mYoloKernel = vYoloKernel;
        mKernelCount = vYoloKernel.size();

        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* 6;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
    }
    YoloLayerPlugin::~YoloLayerPlugin()
    {
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipFree(mAnchor[ii]));
        }
        CUDA_CHECK(hipHostFree(mAnchor));
    }

    int YoloLayerPlugin::getNbOutputs() const noexcept
    {
        return 1;
    }

    YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        read(d, mYoloV5NetWidth);
        read(d, mYoloV5NetHeight);
        read(d, mMaxOutObject);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(mYoloKernel.data(), d, kernelSize);
        d += kernelSize;
        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float) * 6;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
        assert(d == a + length);
    }

    void YoloLayerPlugin::serialize(void* buffer) const noexcept
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        write(d, mYoloV5NetWidth);
        write(d, mYoloV5NetHeight);
        write(d, mMaxOutObject);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(d, mYoloKernel.data(), kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }


    size_t YoloLayerPlugin::getSerializationSize() const noexcept
    {
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount) + sizeof(Yolo::YoloKernel) * mYoloKernel.size() + sizeof(mYoloV5NetWidth) + sizeof(mYoloV5NetHeight) + sizeof(mMaxOutObject);
    }

    bool YoloLayerPlugin::supportsFormatCombination(int pos, const PluginTensorDesc *inOut, int nbInputs, int nbOutputs)const noexcept
    {
        return inOut[pos].format == TensorFormat::kLINEAR && inOut[pos].type == DataType::kFLOAT;
    }

    int YoloLayerPlugin::initialize() noexcept
    {
        return 0;
    }

    Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims) noexcept
    {
        int totalsize = mMaxOutObject * sizeof(Detection) / sizeof(float);

        return Dims3(totalsize + 1, 1, 1);
    }

    void YoloLayerPlugin::setPluginNamespace(const char* pluginNamespace) noexcept
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloLayerPlugin::getPluginNamespace() const noexcept
    {
        return mPluginNamespace;
    }

    DataType YoloLayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const noexcept
    {
        return DataType::kFLOAT;
    }

    bool YoloLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const noexcept
    {
        return false;
    }

    bool YoloLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const noexcept
    {
        return false;
    }

    void YoloLayerPlugin::configurePlugin(PluginTensorDesc const* in, int32_t nbInput, PluginTensorDesc const* out, int32_t nbOutput) noexcept
    {
    }

    void YoloLayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator) noexcept
    {
    }

    void YoloLayerPlugin::detachFromContext() noexcept {}

    const char* YoloLayerPlugin::getPluginType() const noexcept
    {
        return "YoloLayer_TRT";
    }

    const char* YoloLayerPlugin::getPluginVersion() const noexcept
    {
        return "1";
    }

    void YoloLayerPlugin::destroy() noexcept
    {
        delete this;
    }

    IPluginV2IOExt* YoloLayerPlugin::clone() const noexcept
    {
        YoloLayerPlugin* p = new YoloLayerPlugin(mClassCount, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, mYoloKernel);
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data) { return 1.0f / (1.0f + expf(-data)); }

    __global__ void CalDetection(const float *input, float *output, int noElements,
        const int netwidth, const int netheight, int maxoutobject, int yoloWidth, int yoloHeight, const float anchors[6], int classes, int outputElem)
    {

        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid * bnIdx;
        int info_len_i = 5 + classes;
        const float* curInput = input + bnIdx * (info_len_i * total_grid * 3);

        for (int k = 0; k < 3; ++k) {
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            if (box_prob < 0.1f) continue;
            int class_id = 0;
            float max_cls_prob = 0.0;
            for (int i = 5; i < info_len_i; ++i) {
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                if (p > max_cls_prob) {
                    max_cls_prob = p;
                    class_id = i - 5;
                }
            }
            float *res_count = output + bnIdx * outputElem;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= maxoutobject) return;
            char *data = (char*)res_count + sizeof(float) + count * sizeof(Detection);
            Detection *det = (Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            det->bbox[0] = (col - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * netwidth / yoloWidth;
            det->bbox[1] = (row - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * netheight / yoloHeight;
            det->bbox[2] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]);
            det->bbox[2] = det->bbox[2] * det->bbox[2] * anchors[2 * k];
            det->bbox[3] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]);
            det->bbox[3] = det->bbox[3] * det->bbox[3] * anchors[2 * k + 1];
            det->conf = box_prob * max_cls_prob;
            det->class_id = class_id;
        }
    }

    void YoloLayerPlugin::forwardGpu(const float* const* inputs, float *output, hipStream_t stream, int batchSize)
    {
        int outputElem = 1 + mMaxOutObject * sizeof(Detection) / sizeof(float);
        for (int idx = 0; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemsetAsync(output + idx * outputElem, 0, sizeof(float), stream));
        }
        int numElem = 0;
        for (unsigned int i = 0; i < mYoloKernel.size(); ++i) {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width * yolo.height * batchSize;
            if (numElem < mThreadCount) mThreadCount = numElem;

            CalDetection << < (numElem + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream >> >
            (inputs[i], output, numElem, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, yolo.width, yolo.height, (float*)mAnchor[i], mClassCount, outputElem);
        }
    }


    int YoloLayerPlugin::enqueue(int batchSize, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
    {
        forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    PluginFieldCollection YoloPluginCreator::mFC{};
    std::vector<PluginField> YoloPluginCreator::mPluginAttributes;

    YoloPluginCreator::YoloPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloPluginCreator::getPluginName() const noexcept
    {
        return "YoloLayer_TRT";
    }

    const char* YoloPluginCreator::getPluginVersion() const noexcept
    {
        return "1";
    }

    const PluginFieldCollection* YoloPluginCreator::getFieldNames() noexcept
    {
        return &mFC;
    }

    IPluginV2IOExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) noexcept
    {
        assert(fc->nbFields == 2);
        assert(strcmp(fc->fields[0].name, "netinfo") == 0);
        assert(strcmp(fc->fields[1].name, "kernels") == 0);
        int *p_netinfo = (int*)(fc->fields[0].data);
        int class_count = p_netinfo[0];
        int input_w = p_netinfo[1];
        int input_h = p_netinfo[2];
        int max_output_object_count = p_netinfo[3];
        std::vector<Yolo::YoloKernel> kernels(fc->fields[1].length);
        memcpy(&kernels[0], fc->fields[1].data, kernels.size() * sizeof(Yolo::YoloKernel));
        YoloLayerPlugin* obj = new YoloLayerPlugin(class_count, input_w, input_h, max_output_object_count, kernels);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) noexcept
    {
        YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }
    void YoloPluginCreator::setPluginNamespace(const char *libNamespace)  noexcept
    {
        mNamespace = libNamespace;
    }
    const char* YoloPluginCreator::getPluginNamespace()const noexcept
    {
        return mNamespace.c_str();
    }
}
